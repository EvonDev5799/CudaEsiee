#include "hip/hip_runtime.h"
#include <stdio.h>
#include <chrono>
#include <utils.hpp>

struct hipComplex {
	float r;
	float i;
	__device__ hipComplex( float a, float b ) : r(a), i(b) {}

	__device__ float magnitude2( void ) {
		return r * r + i * i;
	}

	__device__ hipComplex operator*(const hipComplex& a) {
		return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
	}

	__device__ hipComplex operator+(const hipComplex& a) {
		return hipComplex(r+a.r, i+a.i);
	}
};

__device__ int julia( int x, int y ) {
	const float scale = 1.5;
	float jx = scale * (float)(DIM/2 - x)/(DIM/2);
	float jy = scale * (float)(DIM/2 - y)/(DIM/2);
	hipComplex c(-0.8, 0.156);
	hipComplex a(jx, jy);
	int i = 0;
	for (i=0; i<200; i++) {
		a = a * a + c;
		if (a.magnitude2() > 1000)
			return 0;
	}
	return 1;
}


__global__ void kernel( unsigned char *ptr, int height, int width ) {
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	int size = height * width;
	if(offset < size) {
		int y = offset/width;
		int x = offset - y * width;
		int juliaValue = julia( x, y );
		ptr[offset] = juliaValue;
	}
}

int main( void ) {	
	double time_taken;
	std::chrono::steady_clock::time_point start, end;

	int pixelQuantity = DIM * DIM;
	hipDeviceProp_t prop;
	hipGetDeviceProperties( &prop, 0);
	int capacity = prop.maxThreadsPerBlock;
	int blocQuantity = (pixelQuantity + capacity - 1)/capacity;
	unsigned char * data = (unsigned char*) malloc(pixelQuantity * sizeof(unsigned char));
	unsigned char * dev_data;
	hipMalloc( (void**)&dev_data, pixelQuantity * sizeof(unsigned char));

	start = std::chrono::high_resolution_clock::now();
	kernel<<<blocQuantity,capacity>>>(dev_data, DIM, DIM);
	hipDeviceSynchronize();
	end = std::chrono::high_resolution_clock::now();
	time_taken = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
	time_taken *= 1e-6;
	printf("Time to compute with max thread capacity and multibloc: %.4lf ms\n", time_taken);
	
	start = std::chrono::high_resolution_clock::now();
	hipMemcpy( data, dev_data, DIM * DIM * sizeof(unsigned char), hipMemcpyDeviceToHost );
	hipDeviceSynchronize();
	end = std::chrono::high_resolution_clock::now();
	time_taken = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
	time_taken *= 1e-6;
	printf("Time to transfer data: %.4lf ms\n", time_taken);
	hipFree(dev_data);
	
	start = std::chrono::high_resolution_clock::now();
	rgb_data* pixels = (rgb_data*) malloc(DIM * DIM * sizeof(rgb_data));
	for (int x = 0; x < DIM; x++) {
		for (int y = 0; y < DIM; y++) {
			int a = y * DIM + x;

			if (data[a] == 0) {
				pixels[a] = black();
			} else {
				pixels[a] = white();
			}
		}
	}
	free(data);
	end = std::chrono::high_resolution_clock::now();
	time_taken = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
	time_taken *= 1e-6;
	printf("Time to transform data: %.4lf ms\n", time_taken);
	
	start = std::chrono::high_resolution_clock::now();
	save_bitmap("julia2.bmp", DIM, DIM, 96, pixels);
	free(pixels);
	end = std::chrono::high_resolution_clock::now();
	time_taken = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
	time_taken *= 1e-6;
	printf("Time to save data to file: %.4lf ms\n", time_taken);
}


