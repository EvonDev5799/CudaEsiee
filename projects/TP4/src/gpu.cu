#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <chrono>
#include <utils.hpp>

__global__ void kerneladd(float* dev_input1, float* dev_input2, float* dev_output){
	dev_output[blockIdx.x] = dev_input1[blockIdx.x] + dev_input2[blockIdx.x];
}

int main(void)
{
	float input1[SIZE];
	float input2[SIZE];
	float output[SIZE];	
	setupArray(input1, SIZE, MAX);
	setupArray(input2, SIZE, MAX);

	float *dev_input1,*dev_input2, *dev_output;
	hipMalloc(&dev_input1, sizeof(float) * SIZE);
	hipMalloc(&dev_input2, sizeof(float) * SIZE);
	hipMalloc(&dev_output, sizeof(float) * SIZE);
	hipMemcpy(dev_input1, input1, sizeof(float) * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(dev_input2, input2, sizeof(float) * SIZE, hipMemcpyHostToDevice);
	
	auto start = std::chrono::high_resolution_clock::now();
	kerneladd<<<SIZE,1>>>(dev_input1, dev_input2, dev_output);
	auto end = std::chrono::high_resolution_clock::now();

	hipMemcpy(output, dev_output, sizeof(float) * SIZE, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	#ifdef DEBUG
	displayResults2(input1, input2, output, SIZE);
	#endif
	double time_taken = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
	time_taken *= 1e-6;
	printf("Time to generate: %.4lf ms\n", time_taken);
}