#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <chrono>
#include <utils.hpp>

__global__ void kernelsqrt(float* dev_input, float* dev_output){
	dev_output[blockIdx.x] = sqrtf(dev_input[blockIdx.x]);
}

int main(void)
{
	float input[DIM];
	float output[DIM];
	setupArray(input, DIM, MAX);

	float *dev_input, *dev_output;
	hipMalloc(&dev_input, sizeof(float) * DIM);
	hipMalloc(&dev_output, sizeof(float) * DIM);
	hipMemcpy(dev_input, input, sizeof(float) * DIM, hipMemcpyHostToDevice);	

	auto start = std::chrono::high_resolution_clock::now();
	kernelsqrt<<<DIM,1>>>(dev_input, dev_output);
	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now();

	hipMemcpy(output, dev_output, sizeof(float) * DIM, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	#ifdef DEBUG
	displayResults(input, output, DIM);
	#endif
	double time_taken = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
	time_taken *= 1e-6;
	printf("Time to generate on GPU with multibloc: %.4lf ms\n", time_taken);
	return 0;
}