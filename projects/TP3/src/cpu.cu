#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <chrono>
#include <utils.hpp>

int main(void)
{
	float input[DIM];
	float output[DIM];	
	setupArray(input, DIM, MAX);
	
	auto start = std::chrono::high_resolution_clock::now();
	for (int i = 0; i < DIM; i++)
		output[i]= sqrtf(input[i]);
	auto end = std::chrono::high_resolution_clock::now();
	#ifdef DEBUG
	displayResults(input, output, DIM);
	#endif
	double time_taken = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
	time_taken *= 1e-6;
	printf("Time to generate with CPU: %.4lf ms\n", time_taken);
}